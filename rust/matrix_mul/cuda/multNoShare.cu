#include "hip/hip_runtime.h"
#include "multNoShare.cuh"

#include <stdlib.h>
#include <bsd/stdlib.h>
#include <iostream>

__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) {
    float Cvalue = 0.0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row > A.height || col > B.width) return;

    for (int e = 0; e < A.width; ++e) {
        Cvalue += (A.elements[row * A.width + e]) * (B.elements[e * B.width + col]);
    }
    C.elements[row * C.width + col] = Cvalue;
}

void MatMul(const Matrix A, const Matrix B, Matrix C) {
    Matrix d_A;
    d_A.width = A.width;
    d_A.height = A.height;
    size_t size = A.width * A.height + sizeof(float);
    hipError_t err = hipMalloc(&d_A.elements, size);
    std::cerr << "CUDA malloc A: " << hipGetErrorString(err) << '\n';
    /* printf("CUDA malloc A: %s\n", hipGetErrorString(err)); */
    err = hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
    printf("Copy A to device: %s\n", hipGetErrorString(err));

    Matrix d_B;
    d_B.width = B.width;
    d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    err = hipMalloc(&d_B.elements, size);
    printf("CUDA malloc B:%s\n", hipGetErrorString(err));
    err = hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
    printf("Copy B to device: %s\n", hipGetErrorString(err));

    Matrix d_C;
    d_C.width = C.width;
    d_C.height = C.height;
    size = C.width * C.height + sizeof(float);
    err = hipMalloc(&d_C.elements, size);
    printf("CUDA malloc C:%s\n", hipGetErrorString(err));

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((B.width + dimBlock.x - 1) / dimBlock.x,
                 (A.height + dimBlock.y - 1) / dimBlock.y);

    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
    err = hipDeviceSynchronize();
    printf("Run kernel: %s\n", hipGetErrorString(err));

    err = hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
    printf("Copy C off of the device: %s\n", hipGetErrorString(err));

    hipFree(d_A.elements);
    hipFree(d_B.elements);

}

void print_matrix(Matrix *m) {
    for(int i = 0; i < min(10, m->height); ++i) {
        for(int j = 0; j < min(10, m->width); ++j) {
            printf("%f ", m->elements[i*m->width + j]);
        }
        printf("\n");
    }
}

int main(int argc, char **argv) {
    Matrix A, B, C;
    int a1, a2, b1, b2;
    a1 = atoi(argv[1]);
    a2 = atoi(argv[2]);
    b1 = a2;
    b2 = atoi(argv[3]);

    A.height = a1;
    A.width = a2;
    A.elements = (float*)malloc(A.width * A.height * sizeof(float));
    
    B.height = b1;
    B.width = b2;
    B.elements = (float*)malloc(B.width * B.height * sizeof(float));

    C.height = A.height;
    C.width = B.width;
    C.elements = (float*)malloc(C.width * C.height * sizeof(float));

    for(int i = 0; i < A.height; ++i)
        for(int j = 0; j < A.width; ++j)
            A.elements[i*A.width + j] = (float)(arc4random() % 3);

    for(int i = 0; i < B.height; ++i)
        for(int j = 0; j < B.width; ++j)
            B.elements[i*B.width + j] = (float)(arc4random() % 2);

    MatMul(A, B, C);

    print_matrix(&A);
    print_matrix(&B);
    print_matrix(&C);
}
