#include "hip/hip_runtime.h"
#include "exp_no_share.cuh"


__global__
void MatExpKernel(Matrix<mat_float_t> src, Matrix<mat_float_t> tgt) {
    float_t tgt_val = 0.0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row > src.height || col > src.width) return;

    for (std::size_t i = 0; i < src.width; ++i) {
        tgt_val += (src.elements[row * src.width + i]) 
                 * (src.elements[i * src.width + col]);
    }
    tgt.elements[row * tgt.width + col] = tgt_val;
}

void MatExp(const Matrix<mat_float_t> &src, Matrix<mat_float_t> &tgt) {

    dim3 dimBlock(CUDA_BLOCK_SIZE, CUDA_BLOCK_SIZE);
    dim3 dimGrid((src.width + dimBlock.x - 1) / dimBlock.x,
                 (src.height + dimBlock.y - 1) / dimBlock.y);

    MatExpKernel<<<dimGrid, dimBlock>>>(src, tgt);

}
