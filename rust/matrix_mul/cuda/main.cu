#include "hip/hip_runtime.h"
#include "matrix.h"
#include "utils.cuh"
#include "exp_no_share.cuh"

#include "dot.h"

/* #include <bsd/stdlib.h> */


void print_matrix(Matrix<mat_float_t> &m) {
    for(int i = 0; i < min(10, int(m.height)); ++i) {
        for(int j = 0; j < min(10, int(m.width)); ++j) {
            std::cout << m.elements[i * m.width + j] << ' ';
        }
        std::cout << '\n';
    }
}
/*  */
/* void random_matrix(Matrix<mat_float_t>& m) { */
/*     for (std::size_t i = 0; i < m.height; ++i) { */
/*         for (std::size_t j = 0; j < m.width; ++j) { */
/*             m.elements[i * m.width + j] = (mat_float_t)(arc4random() % 5); */
/*         } */
/*     } */
/* } */


std::size_t get_degree(Matrix<mat_float_t> &m) {
    std::size_t deg = 0;
    for (std::size_t i = 0; i < m.width; ++i)
        if (m.elements[i] != 0.0) ++deg;
    return deg;
}

__global__
void normalize_matrix(Matrix<mat_float_t> m, std::size_t degree) {
    std::size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    std::size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row > m.height || col > m.width) return;
    m.elements[row * m.width + col] /= degree;
}

__global__
void pow2_matrix(const Matrix<mat_float_t> src, Matrix<mat_float_t> tgt) {
    std::size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    std::size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row > src.height || col > src.width) return;
    
    mat_float_t tgt_val = 0.0;
    for (std::size_t i = 0; i < src.width; ++i) {
        tgt_val += (src.elements[row * src.width + i])
                 * (src.elements[i * src.width + col]);
    }
    tgt.elements[row * tgt.width + col] = tgt_val;
}

__global__
void matrix_tvd(const Matrix<mat_float_t> src, double uniform, double* tvd_vector) {

}

double compute_tvd(Matrix<mat_float_t> src, Matrix<mat_float_t> d_src) {
    dim3 dimBlock(CUDA_BLOCK_SIZE, CUDA_BLOCK_SIZE);
    dim3 dimGrid((src.width + dimBlock.x - 1) / dimBlock.x,
                 (src.height + dimBlock.y - 1) / dimBlock.y);
    std::size_t d = get_degree(src);
    normalize_matrix<<<dimGrid, dimBlock>>>(d_src, get_degree(src));
    std::size_t size = src.width * src.height * sizeof(mat_float_t);
    Matrix<mat_float_t> d_swap_mat;
    d_swap_mat.width = src.width;
    d_swap_mat.height = src.height;
    hipError_t err = hipMalloc(&d_swap_mat.elements, size);
    if (err != 0) {
        std::cerr << "Failed to allocate swap matrix!\n";
        std::cerr << hipGetErrorString(err) << '\n';
        std::exit(1);
    }
    pow2_matrix<<<dimGrid, dimBlock>>>(d_src, d_swap_mat);
    return 0.0;
}


int main(int argc, char **argv) {
    Matrix<mat_float_t> src, tgt;
    /* int width, height; */
    /* width = std::atoi(argv[1]); */
    /* height = std::atoi(argv[2]); */
    /*  */
    /* src.width = width; */
    /* src.height = height; */
    /*  */
    /* tgt.width = width; */
    /* tgt.height = height; */
    /*  */
    /* std::size_t size = width * height * sizeof(mat_float_t); */

    /* src.elements = (mat_float_t*)malloc(size); */
    /* tgt.elements = (mat_float_t*)malloc(size); */

    /* random_matrix(src); */

    src = adj_matrix_from_dot(argv[1]);

    std::cout << src.width << "   " << src.height << '\n';
    print_matrix(src);

    std::size_t size = src.width * src.height * sizeof(mat_float_t);

    tgt.width = src.width;
    tgt.height = src.height;
    tgt.elements = (mat_float_t*)malloc(size);

    Matrix<mat_float_t> d_src = copy_matrix_to_device(src);

    compute_tvd(src, d_src);
    /* Matrix<mat_float_t> d_tgt = copy_matrix_to_device(tgt); */

    /* MatExp(d_src, d_tgt); */
    /*  */
    hipMemcpy(src.elements, d_src.elements, size, hipMemcpyDeviceToHost);
    /* hipMemcpy(tgt.elements, d_tgt.elements, size, hipMemcpyDeviceToHost); */

    hipFree(d_src.elements);
    /* hipFree(d_tgt.elements); */

    print_matrix(src);
    /* print_matrix(tgt); */

    free(src.elements);
    free(tgt.elements);
}
